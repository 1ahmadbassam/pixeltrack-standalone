// C++ headers
#include <algorithm>
#include <numeric>
#include <execution>

// CUDA runtime
#include <hip/hip_runtime.h>

// CMSSW headers
#include "CUDACore/cudaCheck.h"
#include "CUDACore/device_unique_ptr.h"
#include "Framework/CountingIterator.h"
#include "plugin-SiPixelClusterizer/SiPixelRawToClusterGPUKernel.h"  // !
#include "plugin-SiPixelClusterizer/gpuClusteringConstants.h"        // !

#include "PixelRecHits.h"
#include "gpuPixelRecHits.h"

namespace pixelgpudetails {

  TrackingRecHit2DCUDA PixelRecHitGPUKernel::makeHitsAsync(SiPixelDigisCUDA const& digis_d,
                                                           SiPixelClustersCUDA const& clusters_d,
                                                           BeamSpotCUDA const& bs_d,
                                                           pixelCPEforGPU::ParamsOnGPU const* cpeParams,
                                                           hipStream_t stream) const {
    auto nHits = clusters_d.nClusters();
    TrackingRecHit2DCUDA hits_d(nHits, cpeParams, clusters_d.clusModuleStart(), stream);

    int threadsPerBlock = 128;
    int blocks = digis_d.nModules();  // active modules (with digis)

#ifdef GPU_DEBUG
    std::cout << "launching getHits kernel for " << blocks << " blocks" << std::endl;
#endif
    if (blocks)  // protect from empty events
      gpuPixelRecHits::getHits<<<blocks, threadsPerBlock, 0, stream>>>(
          cpeParams, bs_d.data(), digis_d.view(), digis_d.nDigis(), clusters_d.view(), hits_d.view());
    cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
    hipDeviceSynchronize();
    cudaCheck(hipGetLastError());
#endif

    // assuming full warp of threads is better than a smaller number...
    if (nHits) {
      auto hitsModuleStart = clusters_d.clusModuleStart();
      auto layerStart = cpeParams->layerGeometry().layerStart;
      auto hitsLayerStart = hits_d.hitsLayerStart();

      std::for_each_n(std::execution::par, counting_iterator{0}, 11, [=](auto i) {
        hitsLayerStart[i] = hitsModuleStart[layerStart[i]];
      });
    }

    if (nHits) {
      cms::cuda::fillManyFromVector(hits_d.phiBinner(), 10, hits_d.iphi(), hits_d.hitsLayerStart(), nHits, 256, stream);
      cudaCheck(hipGetLastError());
    }

#ifdef GPU_DEBUG
    hipDeviceSynchronize();
    cudaCheck(hipGetLastError());
#endif

    return hits_d;
  }

}  // namespace pixelgpudetails
