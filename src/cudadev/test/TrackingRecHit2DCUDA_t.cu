#include "hip/hip_runtime.h"
#include "CUDADataFormats/TrackingRecHit2DHeterogeneous.h"
#include "CUDACore/copyAsync.h"
#include "CUDACore/cudaCheck.h"

namespace testTrackingRecHit2D {

  __global__ void fill(TrackingRecHit2DSOAView* phits) {
    assert(phits);
    assert(phits->nHits() == 200);

    int i = threadIdx.x;
    if (i > 200)
      return;

    // FIXME do something ?
  }

  __global__ void verify(TrackingRecHit2DSOAView const* phits) {
    assert(phits);
    assert(phits->nHits() == 200);

    int i = threadIdx.x;
    if (i > 200)
      return;

    // FIXME do something ?
  }

  void runKernels(TrackingRecHit2DSOAView* hits) {
    assert(hits);
    fill<<<1, 1024>>>(hits);
    verify<<<1, 1024>>>(hits);
  }

}  // namespace testTrackingRecHit2D

int main() {
  hipStream_t stream;
  cudaCheck(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // inner scope to deallocate memory before destroying the stream
  {
    auto nHits = 200;
    TrackingRecHit2DCUDA tkhit(nHits, nullptr, nullptr, stream);

    testTrackingRecHit2D::runKernels(tkhit.view());
  }

  cudaCheck(hipStreamDestroy(stream));

  return 0;
}
